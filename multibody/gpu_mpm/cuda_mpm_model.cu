#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <hip/hip_runtime.h>

#include "multibody/gpu_mpm/cuda_mpm_model.cuh"
#include "multibody/gpu_mpm/cuda_mpm_kernels.cuh"
#include "multibody/gpu_mpm/radix_sort.cuh"

namespace drake {
namespace multibody {
namespace gmpm {

template<typename T>
void GpuMpmState<T>::AddQRCloth(const std::vector<Vec3<T>> &pos,
                                const std::vector<Vec3<T>> &vel,
                                const std::vector<int> &indices) {
    const auto &verts = pos.size();
    const auto &faces = indices.size() / 3;
    assert(faces * 3  == indices.size());

    h_positions_.insert(h_positions_.end(), pos.begin(), pos.end());
    h_velocities_.insert(h_velocities_.end(), vel.begin(), vel.end());

    for (auto &v : indices) {
        h_indices_.push_back(v + n_verts_);
    }

    n_particles_ += verts + faces;
    n_verts_ += verts;
    n_faces_ += faces;
}

template<typename T>
void GpuMpmState<T>::Finalize() {
    h_positions_.resize(n_particles_);
    h_velocities_.resize(n_particles_);

    // NOTE (changyu): at the initial state, position/velocity is organized as [n_faces | n_verts].
    std::move_backward(h_positions_.begin(), h_positions_.begin() + h_positions_.size() - n_faces_, h_positions_.end());
    std::move_backward(h_velocities_.begin(), h_velocities_.begin() + h_velocities_.size() - n_faces_, h_velocities_.end());
    for (auto &v : h_indices_) {
        v += n_faces_;
    }

    // device particle buffer allocation for reorder data
    for (uint32_t i = 0; i < 2; ++i) {
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_positions, sizeof(Vec3<T>) * n_particles_));
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_velocities, sizeof(Vec3<T>) * n_particles_));
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_volumes, sizeof(T) * n_particles_));
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_affine_matrices, sizeof(Mat3<T>) * n_particles_));

        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_pids, sizeof(int) * n_particles_));
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_sort_keys, sizeof(uint32_t) * n_particles_));
        CUDA_SAFE_CALL(hipMalloc(&particle_buffer_[i].d_sort_ids, sizeof(uint32_t) * n_particles_));
        CUDA_SAFE_CALL(hipMemset(particle_buffer_[i].d_sort_keys, 0, sizeof(uint32_t) * n_particles_));
        CUDA_SAFE_CALL(hipMemset(particle_buffer_[i].d_sort_ids, 0, sizeof(uint32_t) * n_particles_));

        if (i == current_particle_buffer_id_) {
            std::vector<int> id_sequence(n_particles_);
            std::iota(id_sequence.begin(), id_sequence.end(), 0);
            CUDA_SAFE_CALL(hipMemcpy(particle_buffer_[i].d_pids, 
                                      id_sequence.data(), 
                                      sizeof(int) * n_particles_, 
                                      hipMemcpyHostToDevice));

            CUDA_SAFE_CALL(hipMemcpy(particle_buffer_[i].d_positions, 
                                      h_positions_.data(), 
                                      sizeof(Vec3<T>) * n_particles_, 
                                      hipMemcpyHostToDevice));
            CUDA_SAFE_CALL(hipMemcpy(particle_buffer_[i].d_velocities, 
                                      h_velocities_.data(), 
                                      sizeof(Vec3<T>) * n_particles_, 
                                      hipMemcpyHostToDevice));
            CUDA_SAFE_CALL(hipMemset(particle_buffer_[i].d_volumes, 0, sizeof(T) * n_particles_));
            CUDA_SAFE_CALL(hipMemset(particle_buffer_[i].d_affine_matrices, 0, sizeof(Mat3<T>) * n_particles_));
        }
    }
    
    // scratch data
    CUDA_SAFE_CALL(hipMalloc(&d_forces_, sizeof(Vec3<T>) * n_particles_));
    CUDA_SAFE_CALL(hipMalloc(&d_taus_, sizeof(Mat3<T>) * n_particles_));
    CUDA_SAFE_CALL(hipMalloc(&d_index_mappings_, sizeof(int) * n_particles_));
    std::vector<int> initial_index_mappings(n_particles_);
    std::iota(initial_index_mappings.begin(), initial_index_mappings.end(), 0);
    CUDA_SAFE_CALL(hipMemcpy(d_index_mappings_, initial_index_mappings.data(), sizeof(int) * n_particles_, hipMemcpyHostToDevice));

    // element-based data
    CUDA_SAFE_CALL(hipMalloc(&d_deformation_gradients_, sizeof(Mat3<T>) * n_faces_));
    CUDA_SAFE_CALL(hipMalloc(&d_Dm_inverses_, sizeof(Mat2<T>) * n_faces_));
    CUDA_SAFE_CALL(hipMalloc(&d_indices_, sizeof(int) * n_faces_ * 3));
    CUDA_SAFE_CALL(hipMemcpy(d_indices_, h_indices_.data(), sizeof(int) * n_faces_ * 3, hipMemcpyHostToDevice));


    // device grid buffer allocation
    // NOTE(changyu): considering the problem size, we pre-allocate the dense grid once and skip the untouched parts when traversal.
    CUDA_SAFE_CALL(hipMalloc(&grid_buffer_.d_g_masses, config::G_DOMAIN_VOLUME * sizeof(T)));
    CUDA_SAFE_CALL(hipMalloc(&grid_buffer_.d_g_momentum, config::G_DOMAIN_VOLUME * sizeof(Vec3<T>)));
    CUDA_SAFE_CALL(hipMalloc(&grid_buffer_.d_g_touched_flags, config::G_GRID_VOLUME * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc(&grid_buffer_.d_g_touched_ids, config::G_GRID_VOLUME * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc(&grid_buffer_.d_g_touched_cnt, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemset(grid_buffer_.d_g_touched_cnt, 0, sizeof(uint32_t)));

    CUDA_SAFE_CALL(hipMalloc(&d_g_Hess_, config::G_DOMAIN_VOLUME * sizeof(Mat3<T>)));
    CUDA_SAFE_CALL(hipMalloc(&d_g_Grad_, config::G_DOMAIN_VOLUME * sizeof(Vec3<T>)));
    CUDA_SAFE_CALL(hipMalloc(&d_g_Dir_, config::G_DOMAIN_VOLUME * sizeof(Vec3<T>)));
    CUDA_SAFE_CALL(hipMalloc(&d_g_alpha_, config::G_DOMAIN_VOLUME * sizeof(T)));
    CUDA_SAFE_CALL(hipMalloc(&d_g_v_star_, config::G_DOMAIN_VOLUME * sizeof(Vec3<T>)));

    radix_sort(this->next_sort_keys(), this->current_sort_keys(), this->next_sort_ids(), this->current_sort_ids(), sort_buffer_, sort_buffer_size_, static_cast<unsigned int>(n_particles_));
    CUDA_SAFE_CALL(hipMalloc(&sort_buffer_, sizeof(unsigned int) * sort_buffer_size_));

    CUDA_SAFE_CALL((
        initialize_fem_state_kernel<<<
        (this->n_faces() + config::DEFAULT_CUDA_BLOCK_SIZE - 1) / config::DEFAULT_CUDA_BLOCK_SIZE, config::DEFAULT_CUDA_BLOCK_SIZE>>>
        (this->n_faces(), this->indices(), this->current_positions(), this->current_velocities(), this->current_volumes(),
         this->deformation_gradients(), this->Dm_inverses())
        ));
}

template<typename T>
void GpuMpmState<T>::Destroy() {
    for (uint32_t i = 0; i < 2; ++i) {
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_positions));
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_velocities));
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_volumes));
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_affine_matrices));

        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_pids));
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_sort_keys));
        CUDA_SAFE_CALL(hipFree(particle_buffer_[i].d_sort_ids));

        // make sure to throw error when illegal access happens
        particle_buffer_[i].d_positions = nullptr;
        particle_buffer_[i].d_velocities = nullptr;
        particle_buffer_[i].d_volumes = nullptr;
        particle_buffer_[i].d_affine_matrices = nullptr;
        particle_buffer_[i].d_pids = nullptr;
        particle_buffer_[i].d_sort_keys = nullptr;
        particle_buffer_[i].d_sort_ids = nullptr;
    }

    CUDA_SAFE_CALL(hipFree(d_forces_));
    CUDA_SAFE_CALL(hipFree(d_taus_));
    CUDA_SAFE_CALL(hipFree(d_index_mappings_));
    CUDA_SAFE_CALL(hipFree(d_deformation_gradients_));
    CUDA_SAFE_CALL(hipFree(d_Dm_inverses_));
    CUDA_SAFE_CALL(hipFree(d_indices_));
    d_forces_ = nullptr;
    d_taus_ = nullptr;
    d_index_mappings_ = nullptr;
    d_deformation_gradients_ = nullptr;
    d_Dm_inverses_ = nullptr;
    d_indices_ = nullptr;

    CUDA_SAFE_CALL(hipFree(grid_buffer_.d_g_masses));
    CUDA_SAFE_CALL(hipFree(grid_buffer_.d_g_momentum));
    CUDA_SAFE_CALL(hipFree(grid_buffer_.d_g_touched_flags));
    CUDA_SAFE_CALL(hipFree(grid_buffer_.d_g_touched_ids));
    CUDA_SAFE_CALL(hipFree(grid_buffer_.d_g_touched_cnt));
    grid_buffer_.d_g_masses = nullptr;
    grid_buffer_.d_g_momentum = nullptr;
    grid_buffer_.d_g_touched_flags = nullptr;
    grid_buffer_.d_g_touched_ids = nullptr;
    grid_buffer_.d_g_touched_cnt = nullptr;

    CUDA_SAFE_CALL(hipFree(d_g_Hess_));
    CUDA_SAFE_CALL(hipFree(d_g_Grad_));
    CUDA_SAFE_CALL(hipFree(d_g_Dir_));
    CUDA_SAFE_CALL(hipFree(d_g_alpha_));
    CUDA_SAFE_CALL(hipFree(d_g_v_star_));
    d_g_Hess_ = nullptr;
    d_g_Grad_ = nullptr;
    d_g_Dir_ = nullptr;
    d_g_alpha_ = nullptr;
    d_g_v_star_ = nullptr;

    CUDA_SAFE_CALL(hipFree(sort_buffer_));
    sort_buffer_ = nullptr;
    sort_buffer_size_ = 0;

    if (d_contact_mpm_id_) {
        CUDA_SAFE_CALL(hipFree(d_contact_mpm_id_));
        d_contact_mpm_id_ = nullptr;
    }
    if (d_contact_pos_) {
        CUDA_SAFE_CALL(hipFree(d_contact_pos_));
        d_contact_pos_ = nullptr;
    }
    if (d_contact_vel_) {
        CUDA_SAFE_CALL(hipFree(d_contact_vel_));
        d_contact_vel_ = nullptr;
    }
    if (d_contact_dist_) {
        CUDA_SAFE_CALL(hipFree(d_contact_dist_));
        d_contact_dist_ = nullptr;
    }
    if (d_contact_normal_) {
        CUDA_SAFE_CALL(hipFree(d_contact_normal_));
        d_contact_normal_ = nullptr;
    }
    if (d_contact_rigid_v_) {
        CUDA_SAFE_CALL(hipFree(d_contact_rigid_v_));
        d_contact_rigid_v_ = nullptr;
    }
    if (d_contact_sort_keys_) {
        CUDA_SAFE_CALL(hipFree(d_contact_sort_keys_));
        d_contact_sort_keys_ = nullptr;
    }
    if (d_contact_sort_ids_) {
        CUDA_SAFE_CALL(hipFree(d_contact_sort_ids_));
        d_contact_sort_keys_ = nullptr;
    }
}

template<typename T>
GpuMpmState<T>::DumpT GpuMpmState<T>::DumpCpuState() const {
    std::vector<Vec3<T>> export_pos;
    std::vector<Vec3<T>> export_original_pos;
    std::vector<int> export_pid;
    std::vector<int> export_indices;
    export_pos.resize(n_particles());
    export_original_pos.resize(n_particles());
    export_pid.resize(n_particles());
    export_indices.resize(n_faces() * 3);
    CUDA_SAFE_CALL(hipMemcpy(export_pos.data(), current_positions(), sizeof(Vec3<T>) * n_particles(), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(export_pid.data(), current_pids(), sizeof(int) * n_particles(), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(export_indices.data(), indices(), sizeof(int) * n_faces() * 3, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < n_particles(); ++i) {
      export_original_pos[export_pid[i]] = export_pos[i];
    }
    export_pos = std::vector<Vec3<T>>(export_original_pos.begin() + n_faces(), export_original_pos.end());
    for (size_t i = 0; i < n_faces() * 3; ++i) {
      export_indices[i] -= n_faces();
    }
    return std::make_tuple(export_pos, export_indices);
}

template<typename T>
void GpuMpmState<T>::ReallocateContacts(size_t num_contacts) {
    this->num_contacts_ = num_contacts;
    if (num_contacts > contact_buffer_size) {
        contact_buffer_size = num_contacts;
        if (d_contact_mpm_id_) {
            CUDA_SAFE_CALL(hipFree(d_contact_mpm_id_));
        }
        if (d_contact_pos_) {
            CUDA_SAFE_CALL(hipFree(d_contact_pos_));
        }
        if (d_contact_vel_) {
            CUDA_SAFE_CALL(hipFree(d_contact_vel_));
        }
        if (d_contact_dist_) {
            CUDA_SAFE_CALL(hipFree(d_contact_dist_));
        }
        if (d_contact_normal_) {
            CUDA_SAFE_CALL(hipFree(d_contact_normal_));
        }
        if (d_contact_rigid_v_) {
            CUDA_SAFE_CALL(hipFree(d_contact_rigid_v_));
        }
        if (d_contact_sort_keys_) {
            CUDA_SAFE_CALL(hipFree(d_contact_sort_keys_));
        }
        if (d_contact_sort_ids_) {
            CUDA_SAFE_CALL(hipFree(d_contact_sort_ids_));
        }
        hipMalloc(&d_contact_mpm_id_, sizeof(uint32_t) * contact_buffer_size);
        hipMalloc(&d_contact_pos_, sizeof(T) * 3 * contact_buffer_size);
        hipMalloc(&d_contact_vel_, sizeof(T) * 3 * contact_buffer_size);
        hipMalloc(&d_contact_dist_, sizeof(T) * contact_buffer_size);
        hipMalloc(&d_contact_normal_, sizeof(T) * 3 * contact_buffer_size);
        hipMalloc(&d_contact_rigid_v_, sizeof(T) * 3 * contact_buffer_size);
        hipMalloc(&d_contact_sort_keys_, sizeof(uint32_t) * contact_buffer_size);
        hipMalloc(&d_contact_sort_ids_, sizeof(uint32_t) * contact_buffer_size);
    }
}

template class GpuMpmState<config::GpuT>;

}  // namespace gmpm
}  // namespace multibody
}  // namespace drake