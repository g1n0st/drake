#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "multibody/gpu_mpm/cuda_mpm_solver.cuh"
#include "multibody/gpu_mpm/cuda_mpm_kernels.cuh"
#include "multibody/gpu_mpm/radix_sort.cuh"

namespace drake {
namespace multibody {
namespace gmpm {

template<typename T>
void GpuMpmSolver<T>::RebuildMapping(GpuMpmState<T> *state) {
    compute_base_cell_node_index<<<(state->n_particles() + config::DEFAULT_CUDA_BLOCK_SIZE - 1) / config::DEFAULT_CUDA_BLOCK_SIZE, config::DEFAULT_CUDA_BLOCK_SIZE>>>(state->n_particles(), state->current_positions(), state->current_sort_keys(), state->current_sort_ids());
}

template class GpuMpmSolver<double>;
template class GpuMpmSolver<float>;

}  // namespace gmpm
}  // namespace multibody
}  // namespace drake